// includes, system

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

// Here you can set the device ID that was assigned to you
#define MYDEVICE 0

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char *msg);

///////////////////////////////////////////////////////////////////////////////
// Program main
///////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    hipSetDevice(MYDEVICE);
    // pointer and dimension for host memory
    int n, dimA;
    float *h_a;

    // pointers for device memory
    float *d_a, *d_b;

    // allocate and initialize host memory
    // Bonus: try using cudaMallocHost in place of malloc
    dimA = 8;
    h_a = (float *) malloc(dimA*sizeof(float));
    for (n=0; n<dimA; n++)
    {
        h_a[n] = (float) n;
    }

    // Part 1 of 5: allocate device memory
    size_t memSize = dimA*sizeof(float);
    hipMalloc((void **)&d_a, memSize);
    hipMalloc((void **)&d_b, memSize);

    // Part 2 of 5: host to device memory copy
    hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);

    // Part 3 of 5: device to device memory copy
    hipMemcpy(d_b, d_a,  memSize, hipMemcpyDeviceToDevice);

    // clear host memory
    for (n=0; n<dimA; n++)
    {
	    h_a[n] = 0.f;
    }

    // Part 4 of 5: device to host copy
    hipMemcpy(h_a, d_a ,memSize, hipMemcpyDeviceToHost);

    // Check for any CUDA errors
    checkCUDAError("hipMemcpy calls");

    // verify the data on the host is correct
    for (n=0; n<dimA; n++)
    {
	    assert(h_a[n] == (float) n);
    }

    // Part 5 of 5: free device memory pointers d_a and d_b
    hipFree(d_a);
    hipFree(d_b);

    // Check for any CUDA errors
    checkCUDAError("hipFree");

    // free host memory pointer h_a
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) 
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
		exit(-1);
	}                         
}
